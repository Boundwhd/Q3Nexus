#include "hip/hip_runtime.h"
#include "../include/common.cuh"
#include "../include/linear.cuh"
#include <hipblas.h>

#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    }

#define CUBLAS_CHECK(err) \
    if (err != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS Error %s:%d: %d\n", __FILE__, __LINE__, err); \
        exit(EXIT_FAILURE); \
    }

// y = xA^T + b
/**
 * x [M, K]
 * A [N, K]
 * y [M, N]
 * custom for Qwen3, no bias
 */
void launch_linear_bf16xbf16(
   const __hip_bfloat16* x,
   const __hip_bfloat16* A,
   __hip_bfloat16* y,
   const int M,
   const int N,
   const int K
) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    const float alpha = 1.0f, beta = 0.0f;
    CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
    CUBLAS_CHECK(cublasSgemmEx(
        handle,
        HIPBLAS_OP_T, HIPBLAS_OP_N,
        N,
        M,
        K,
        &alpha,
        A,
        HIP_R_16BF,
        K,
        x,
        HIP_R_16BF,
        K,
        &beta,
        y,
        HIP_R_16BF,
        N
    ));
    CUBLAS_CHECK(hipblasDestroy(handle));
}